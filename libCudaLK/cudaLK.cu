#include "hip/hip_runtime.h"
/*
@author: Benjamin Lefaudeux (blefaudeux at github)

This program computes optical flow using the nVidia CUDA API

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

DISCLAIMER (Benjamin Lefaudeux)

Some parts of the code below are from Nghia Ho (http://nghiaho.com/)
See nghiaho12 @ yahoo.com (available without specific licence)
*/


#include "cudaLK.h"
#include <stdio.h>

#include "derivativeKernels.cu"
#include "trackingKernels.cu"

const float scaling[] = {1, 0.5f, 0.25f, 0.125f, 0.0625f, 0.03125f, 0.015625f, 0.0078125f};

// TODO :
// - constant velocity model : keep previous velocity if the point was already tracked
// - adaptative gain  ?

// ----------------------------------------------------------------------



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
 *       1  4  6  4  1
 *       4 16 24 16  4
 *       6 24 36 24  6
 *       4 16 24 16  4
 *       1  4  6  4  1
 */

// Compute clock time
timespec cudaLK::diffTime(timespec start, timespec end)
{
    timespec temp;
    if ((end.tv_nsec-start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec-start.tv_sec;
        temp.tv_nsec = end.tv_nsec-start.tv_nsec;
    }
    return temp;
}

// Compute divUp / Useful for kernel job repartition
int iDivUp( int a,  int b )
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}




cudaLK::cudaLK()
{
    // Use default values for pyramid levels & LK search radius
    _n_pyramids     = LEVELS;
    _patch_radius   = PATCH_R;
    _max_points     = MAX_POINTS;
    _n_threads_x    = NTHREAD_X;
    _n_threads_y    = NTHREAD_Y;

    hipMemcpyToSymbol(HIP_SYMBOL(LK_patch), &_patch_radius, sizeof(int));   // in device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(LK_points), &_max_points, sizeof(int));

    // Init flags
    b_mem_allocated   = false;
    b_mem4_allocated  = false;
    b_first_time      = true;
    b_use_weighted_norm = false;
}

cudaLK::cudaLK( int n_pyramids,
                int patch_radius ,
                int n_max_points,
                bool weighted_norm )
{
    //  /!\ Memory is not allocated at this point
    // Call initMem() or initMem4Frame()

    // Specify pyramid levels and LK search radius
    _n_pyramids     = n_pyramids;
    _patch_radius   = patch_radius;
    _max_points     = n_max_points;
    _n_threads_x    = NTHREAD_X;
    _n_threads_y    = NTHREAD_Y;

    hipMemcpyToSymbol(HIP_SYMBOL(LK_patch), &_patch_radius, sizeof(int));   // in device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(LK_points), &_max_points, sizeof(int));


    // Init flags
    b_mem_allocated   = false;
    b_mem4_allocated  = false;
    b_first_time      = true;
    b_use_weighted_norm = weighted_norm;
}

cudaLK::~cudaLK()
{
    releaseMem ();
}

void cudaLK::bindTextureUnits( hipArray *pict0,
                               hipArray *pict1,
                               hipArray *deriv_x,
                               hipArray *deriv_y )
{

    hipUnbindTexture (gpu_textr_pict_0);
    hipUnbindTexture (gpu_textr_pict_1);
    hipUnbindTexture (gpu_textr_deriv_x);
    hipUnbindTexture (gpu_textr_deriv_y);

    hipBindTextureToArray(gpu_textr_pict_0,  pict0, gpu_textr_pict_0.channelDesc);
    hipBindTextureToArray(gpu_textr_pict_1,  pict1, gpu_textr_pict_1.channelDesc);

    hipBindTextureToArray (gpu_textr_deriv_x,   deriv_x, gpu_textr_deriv_x.channelDesc);
    hipBindTextureToArray (gpu_textr_deriv_y,   deriv_y, gpu_textr_deriv_y.channelDesc);
}

void cudaLK::buildPyramids()
{
    // 1D & 2D-indexing of kernels
    int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
    int blocksH = h/_n_threads_y + ((h % _n_threads_y )?1:0);
    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Build pyramids
    for(int i=0; i < _n_pyramids-1; i++) {
        kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_prev1[i], pyr_w[i], pyr_h[i], gpu_smoothed_prev1_x);
        kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_prev2[i], pyr_w[i], pyr_h[i], gpu_smoothed_prev2_x);

        kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_prev1_x, pyr_w[i], pyr_h[i], gpu_smoothed_prev1);
        kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_prev2_x, pyr_w[i], pyr_h[i], gpu_smoothed_prev2);

        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_prev1, pyr_w[i], pyr_h[i], gpu_img_pyramid_prev1[i+1], pyr_w[i+1], pyr_h[i+1]);
        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_prev2, pyr_w[i], pyr_h[i], gpu_img_pyramid_prev2[i+1], pyr_w[i+1], pyr_h[i+1]);
    }
}

void cudaLK::checkCUDAError(const char *msg) {
    // Check GPU status to catch errors
    // "msg" is printed in case of an exception

    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


void cudaLK::computeDerivatives(float const *in,
                                float       *deriv_buff_x,
                                float       *deriv_buff_y,
                                int         pyr_level,
                                hipArray   *gpu_array_deriv_x,
                                hipArray   *gpu_array_deriv_y) {
    //
    // Compute spatial derivatives using Scharr or Sobel operator
    //

    // 2D-indexing for kernel managements
    int _n_threads_x = NTHREAD_X;
    int _n_threads_y = NTHREAD_Y;
    int _w = pyr_w[pyr_level], _h=pyr_h[pyr_level];

    int blocksW = _w/_n_threads_x + ((_w % _n_threads_x)?1:0);
    int blocksH = _h/_n_threads_y + ((_h % _n_threads_y)?1:0);

    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Compute via kernels
    kernelScharrX <<<blocks, threads>>>(in, _w, _h, deriv_buff_x);
    kernelScharrY <<<blocks, threads>>>(in, _w, _h, deriv_buff_y);

    checkCUDAError("ComputingSpatialDerivatives");

    // Copy result to texture buffers
    int offset = 0;

    for (int i=0; i<pyr_level; ++i)
    {
        offset += pyr_w[i];
    }

    hipMemcpy2DToArrayAsync(gpu_array_deriv_x,
                             offset * sizeof(float),
                             0,
                             deriv_buff_x,
                             sizeof(float)*_w,
                             sizeof(float)*_w,
                             _h,
                             hipMemcpyDeviceToDevice);

    hipMemcpy2DToArrayAsync(gpu_array_deriv_y,
                             offset * sizeof(float),
                             0,
                             deriv_buff_y,
                             sizeof(float)*_w,
                             sizeof(float)*_w,
                             _h,
                             hipMemcpyDeviceToDevice);

    //  checkCUDAError("ComputingSpatialDerivatives-memdump");
}

void cudaLK::cvtPicture(bool useCurrent, bool cvtToGrey)
{
    int blocks1D = (w*h)/256 + (w*h % 256?1:0); // for greyscale

    if (useCurrent)
    {
        if (cvtToGrey) {
            // RGB -> grey
            convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_cur1_RGB, gpu_img_pyramid_cur1[0], w*h);
            convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_cur2_RGB, gpu_img_pyramid_cur2[0], w*h);
        } else {
            convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_cur1_RGB, gpu_img_pyramid_cur1[0], w*h);
            convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_cur2_RGB, gpu_img_pyramid_cur2[0], w*h);
        }
    }
    else
    {
        if (cvtToGrey) {
            // RGB -> grey
            convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_prev1_RGB, gpu_img_pyramid_prev1[0], w*h);
            convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_prev2_RGB, gpu_img_pyramid_prev2[0], w*h);
            checkCUDAError("convertRGBToGrey");
        } else {
            convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_prev1_RGB, gpu_img_pyramid_prev1[0], w*h);
            convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_prev2_RGB, gpu_img_pyramid_prev2[0], w*h);
            checkCUDAError("convertToFloat");
        }
    }
}

void cudaLK::exportDebug(IplImage *outPict) {
    // Debug function to see what's going on in picture buffers
    // Not reliable for IPLImages because of widthStep --> TODO ?

    // Copy buffer back to host
    float pict_x_f[w*h];
    //  float pict_y_f[w*h];

    // SOBEL
    hipMemcpy (pict_x_f, gpu_img_pyramid_prev1[0], w*h*sizeof(float), hipMemcpyDeviceToHost);
    //  hipMemcpy (pict_x_f, gpu_img_pyramid_cur1[0], w*h*sizeof(float), hipMemcpyDeviceToHost);

    // Get picture max value
    float val = 0.f;
    float max_val = 0.f;
    for (int i = 0; i<w; ++i) {
        for (int j=0; j<h; ++j) {
            val = pict_x_f[i +j*w];
            if (val > max_val)
                max_val = val;
        }
    }

    // Convert to char
    for (int i = 0; i<w; ++i) {
        for (int j=0; j<h; ++j) {
            val = pict_x_f[i +j*w];
            outPict->imageData[i +j*outPict->widthStep] = (unsigned char) round(val/max_val*254);
        }
    }

    // Check derivatives

    //  hipMemcpy(pict_x_f, gpu_deriv_x, w*h*sizeof(float), hipMemcpyDeviceToHost);
    //  hipMemcpy(pict_y_f, gpu_deriv_y, w*h*sizeof(float), hipMemcpyDeviceToHost);
    //  checkCUDAError ("ExportDebugFunction");

    //  // Get picture max value
    //  float val = 0.f;
    //  float max_val = 0.f;
    //  for (int i = 0; i<w; ++i) {
    //    for (int j=0; j<h; ++j) {
    //      val = sqrt(pict_x_f[i +j*w]*pict_x_f[i +j*w] + pict_y_f[i +j*w]*pict_y_f[i +j*w]);
    ////      val = pict_y_f[i +j*w];
    //      if (val > max_val)
    //        max_val = val;
    //    }

    //  }

    //  printf("ExportDebug max value : %d x %d - %f\n", w, h, max_val);

    //  // Convert to char
    //  for (int i = 0; i<w; ++i) {
    //    for (int j=0; j<h; ++j) {
    //      val = sqrt(pict_x_f[i +j*w]*pict_x_f[i +j*w] + pict_y_f[i +j*w]*pict_y_f[i +j*w]);
    ////      val = pict_y_f[i +j*w];
    //      outPict->imageData[i +j*outPict->widthStep] = (unsigned char) round(val/max_val*254);
    //    }
    //  }

    checkCUDAError ("Debug exportation");
}

void cudaLK::initMem()
{
    // Picture buffers
    hipMalloc((void**)&gpu_img_prev_RGB, sizeof(char)*w*h*3);
    hipMalloc((void**)&gpu_img_cur_RGB, sizeof(char)*w*h*3);

    hipMalloc((void**)&gpu_img_pyramid_prev1[0], sizeof(float)*w*h);
    hipMalloc((void**)&gpu_img_pyramid_cur1[0], sizeof(float)*w*h);

    hipMalloc((void**)&gpu_smoothed_prev_x, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_smoothed_cur_x, sizeof(float)*w*h);

    hipMalloc((void**)&gpu_smoothed_prev, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_smoothed_cur, sizeof(float)*w*h);

    // Indexes
    hipMalloc ((void**) &gpu_pt_indexes, 2*MAX_POINTS*sizeof(float));

    // Texture
    hipMallocArray(&gpu_array_pyramid_prev, &texRef_pyramid_prev.channelDesc, w, h);
    hipMallocArray(&gpu_array_pyramid_cur, &texRef_pyramid_cur.channelDesc, w, h);

    hipBindTextureToArray(texRef_pyramid_prev, gpu_array_pyramid_prev, texRef_pyramid_prev.channelDesc);
    hipBindTextureToArray(texRef_pyramid_cur,  gpu_array_pyramid_cur,  texRef_pyramid_cur.channelDesc);

    texRef_pyramid_prev.normalized = 0;
    texRef_pyramid_prev.filterMode = hipFilterModeLinear;
    texRef_pyramid_prev.addressMode[0] = hipAddressModeClamp;
    texRef_pyramid_prev.addressMode[1] = hipAddressModeClamp;

    texRef_pyramid_cur.normalized = 0;
    texRef_pyramid_cur.filterMode = hipFilterModeLinear;
    texRef_pyramid_cur.addressMode[0] = hipAddressModeClamp;
    texRef_pyramid_cur.addressMode[1] = hipAddressModeClamp;

    hipMalloc((void**)&gpu_dx, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_dy, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_status, sizeof(char)*w*h);

    int _w = w;
    int _h = h;

    dx1 = new float[w*h];
    dy1 = new float[w*h];
    status = new char[w*h];

    pyr_w[0] = w;
    pyr_h[0] = h;

    for(int i=1; i < _n_pyramids; ++i)
    {
        _w /= 2;
        _h /= 2;
        pyr_w[i] = _w;
        pyr_h[i] = _h;

        hipMalloc((void**)&gpu_img_pyramid_prev1[i], sizeof(float)*_w*_h);
        hipMalloc((void**)&gpu_img_pyramid_cur1[i], sizeof(float)*_w*_h);
    }

    b_mem_allocated = true;
    printf("[CudaKLT]: Memory allocated\n");
}

void cudaLK::initMem4Frame()
{
    // Allocate picture buffers
    hipMalloc((void**)&gpu_img_prev1_RGB,  sizeof(char) * w * h * 3);
    hipMalloc((void**)&gpu_img_prev2_RGB,  sizeof(char) * w * h * 3);
    hipMalloc((void**)&gpu_img_cur1_RGB,   sizeof(char) * w * h * 3);
    hipMalloc((void**)&gpu_img_cur2_RGB,   sizeof(char) * w * h * 3);

    // Allocate Pyramids
    hipMalloc((void**)&gpu_img_pyramid_prev1[0], sizeof(float) * w * h);
    hipMalloc((void**)&gpu_img_pyramid_prev2[0], sizeof(float) * w * h);
    hipMalloc((void**)&gpu_img_pyramid_cur1[0],  sizeof(float) * w * h);
    hipMalloc((void**)&gpu_img_pyramid_cur2[0],  sizeof(float) * w * h);

    // Allocate smoothed pictures (for pyramid building)
    hipMalloc((void**)&gpu_smoothed_prev1_x, sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_prev2_x, sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_cur1_x,  sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_cur2_x,  sizeof(float) * w * h);

    hipMalloc((void**)&gpu_smoothed_prev1, sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_prev2, sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_cur1 , sizeof(float) * w * h);
    hipMalloc((void**)&gpu_smoothed_cur2 , sizeof(float) * w * h);

    // Allocate spatial derivatives & pict buffer
    hipMalloc((void**)&gpu_deriv_x,   sizeof(float) * w * h);
    hipMalloc((void**)&gpu_deriv_y,   sizeof(float) * w * h);

    // Allocate LK compute intermediates :
    hipMalloc((void **) &gpu_neighbourhood_det, sizeof(float) * MAX_POINTS);
    hipMalloc((void **) &gpu_neighbourhood_Iyy, sizeof(float) * MAX_POINTS);
    hipMalloc((void **) &gpu_neighbourhood_Ixy, sizeof(float) * MAX_POINTS);
    hipMalloc((void **) &gpu_neighbourhood_Ixx, sizeof(float) * MAX_POINTS);

    // Indexes
    hipMalloc ((void**) &gpu_pt_indexes , 2 * MAX_POINTS * sizeof(float));

    // Textures - Keep all the gradients in texture space, everytime !
    hipMallocArray(&gpu_array_pict_0, &gpu_textr_pict_0.channelDesc, 2 * w, h);
    hipMallocArray(&gpu_array_pict_1, &gpu_textr_pict_0.channelDesc, 2 * w, h);
    hipMallocArray(&gpu_array_pict_2, &gpu_textr_pict_0.channelDesc, 2 * w, h);
    hipMallocArray(&gpu_array_pict_3, &gpu_textr_pict_0.channelDesc, 2 * w, h);

    hipMallocArray(&gpu_array_deriv_x_0, &gpu_textr_deriv_x.channelDesc, 2 * w,h); // the array will receive all the pyramid levels !
    hipMallocArray(&gpu_array_deriv_y_0, &gpu_textr_deriv_y.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_x_1, &gpu_textr_deriv_x.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_y_1, &gpu_textr_deriv_y.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_x_2, &gpu_textr_deriv_x.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_y_2, &gpu_textr_deriv_y.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_x_3, &gpu_textr_deriv_x.channelDesc, 2 * w,h);
    hipMallocArray(&gpu_array_deriv_y_3, &gpu_textr_deriv_y.channelDesc, 2 * w,h);

    setupTextures();

    // Displacements
    hipMalloc((void**)&gpu_dx, sizeof(float) * MAX_POINTS);
    hipMalloc((void**)&gpu_dy, sizeof(float) * MAX_POINTS);

    hipMalloc((void**)&gpu_dx1, sizeof(float) * MAX_POINTS);
    hipMalloc((void**)&gpu_dy1, sizeof(float) * MAX_POINTS);

    hipMalloc((void**)&gpu_dx2, sizeof(float) * MAX_POINTS);
    hipMalloc((void**)&gpu_dy2, sizeof(float) * MAX_POINTS);

    hipMalloc((void**)&gpu_dx3, sizeof(float) * MAX_POINTS);
    hipMalloc((void**)&gpu_dy3, sizeof(float) * MAX_POINTS);


    // Check GPU status
    hipMalloc((void**)&gpu_status, sizeof(char) * MAX_POINTS);

    // Pyramids
    int _w = w;
    int _h = h;

    // Allocate pinned memory on host
    hipHostAlloc((void**)&dx1, MAX_POINTS * sizeof(float), 0);
    hipHostAlloc((void**)&dy1, MAX_POINTS * sizeof(float), 0);

    hipHostAlloc((void**)&dx2, MAX_POINTS * sizeof(float), 0);
    hipHostAlloc((void**)&dy2, MAX_POINTS * sizeof(float), 0);

    hipHostAlloc((void**)&dx3, MAX_POINTS * sizeof(float), 0);
    hipHostAlloc((void**)&dy3, MAX_POINTS * sizeof(float), 0);

    hipHostAlloc((void**)&dx4, MAX_POINTS * sizeof(float), 0);
    hipHostAlloc((void**)&dy4, MAX_POINTS * sizeof(float), 0);

    hipHostAlloc((void**)&status, MAX_POINTS * sizeof(char), 0);

    checkCUDAError ("Memory Allocation");

    pyr_w[0] = w;
    pyr_h[0] = h;

    for(int i=1; i < _n_pyramids; ++i) {
        _w /= 2;
        _h /= 2;
        pyr_w[i] = _w;  // Pyramid size
        pyr_h[i] = _h;

        hipMalloc((void**)&gpu_img_pyramid_prev1[i], sizeof(float)*_w*_h);
        hipMalloc((void**)&gpu_img_pyramid_prev2[i], sizeof(float)*_w*_h);
        hipMalloc((void**)&gpu_img_pyramid_cur1[i] , sizeof(float)*_w*_h);
        hipMalloc((void**)&gpu_img_pyramid_cur2[i] , sizeof(float)*_w*_h);
    }

    // That's all, folks
    this->b_mem4_allocated = true;

    checkCUDAError ("Allocating 4Frames memory");
    printf("[CucaKLT] : 4Frames memory allocated\n");
}

void cudaLK::dummyCall()
{
    // Do something on the GPU to wake up the beast..
    int dummy = 1;
    hipMemcpyToSymbol(HIP_SYMBOL(LK_width), &dummy, sizeof(int));
}

void cudaLK::fillDerivatives(float **pict_pyramid,
                             hipArray *gpu_array_deriv_x,
                             hipArray *gpu_array_deriv_y)
{
    // Compute derivatives & load them into texture units
    for(int l = _n_pyramids-1; l >= 0; l--) {
        computeDerivatives(pict_pyramid[l],
                           gpu_deriv_x,       // Buffers
                           gpu_deriv_y,
                           l,
                           gpu_array_deriv_x, // Final texture recipients
                           gpu_array_deriv_y);
    }
}

void cudaLK::loadBackPictures(const IplImage *prev1,
                              const IplImage *prev2,
                              bool b_CvtToGrey) {

    // Load initial pictures to be used in backbuffers (and allocate memory if nedded)
    // Just called once

    // Allocate memory if needed
    if (!b_mem4_allocated) {
        w = prev1->width;
        h = prev1->height;

        // Initiate constant memory variables
        hipMemcpyToSymbol(HIP_SYMBOL(LK_width), &w, sizeof(w));
        hipMemcpyToSymbol(HIP_SYMBOL(LK_height), &h, sizeof(h));

        initMem4Frame ();

        int n_iterations  = MAX_ITERATIONS;
        float threshold   = MV_THRESHOLD;
        hipMemcpyToSymbol(HIP_SYMBOL(LK_iteration), &n_iterations, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(LK_threshold), &threshold, sizeof(float));

        // Init weighting parameters, if needed :
        float temp_weight_array[MAX_WEIGHT_VALUES];

        if (w*h > MAX_WEIGHT_VALUES) {
            // Window is too big.. no weighting for now
            this->b_use_weighted_norm = false;
        }

        if (this->b_use_weighted_norm) {
            for (int i = -w; i<= w; ++i) {
                for (int j = -h; j<= h; ++j) {
                    temp_weight_array[i + j*w] = exp (-(i*j)/10.f); // TODO : handle std settings gracefully..
                }
            }

            hipMemcpyToSymbol(HIP_SYMBOL(LK_Weight), &temp_weight_array, w*h*sizeof(float), hipMemcpyHostToDevice);
        }
    }
    checkCUDAError("LoadBackPicture - set symbols");

    // Transfer from host memspace to gpu memspace
    if (b_CvtToGrey) {
        hipMemcpy2D (gpu_img_prev1_RGB, w*sizeof(uchar), prev1->imageData, prev1->widthStep, 3 * prev1->width * sizeof(uchar), prev1->height, hipMemcpyHostToDevice );
        hipMemcpy2D (gpu_img_prev2_RGB, w*sizeof(uchar), prev2->imageData, prev2->widthStep, 3 * prev2->width * sizeof(uchar), prev2->height, hipMemcpyHostToDevice );
    } else {
        hipMemcpy2D (gpu_img_prev1_RGB, w*sizeof(uchar), prev1->imageData, prev1->widthStep, prev1->width * sizeof(uchar), prev1->height, hipMemcpyHostToDevice );
        hipMemcpy2D (gpu_img_prev2_RGB, w*sizeof(uchar), prev2->imageData, prev2->widthStep, prev2->width * sizeof(uchar), prev2->height, hipMemcpyHostToDevice );
    }
    checkCUDAError("LoadBackPicture");


    // Convert picture to floats & grey
    cvtPicture(false, b_CvtToGrey);
    buildPyramids();
    checkCUDAError("pyrDownsample");

    // Load hipArray buffer from pyramids
    int pyr_offset = 0;
    for (int l=0; l<_n_pyramids; ++l) {
        hipMemcpy2DToArrayAsync (gpu_array_pict_0,
                                  pyr_offset * sizeof(float),
                                  0,
                                  gpu_img_pyramid_prev1[l],
                                  sizeof(float)*pyr_w[l],
                                  sizeof(float)*pyr_w[l],
                                  pyr_h[l],
                                  hipMemcpyDeviceToDevice);

        hipMemcpy2DToArrayAsync(gpu_array_pict_1,
                                 pyr_offset * sizeof(float),
                                 0,
                                 gpu_img_pyramid_prev2[l],
                                 sizeof(float)*pyr_w[l],
                                 sizeof(float)*pyr_w[l],
                                 pyr_h[l],
                                 hipMemcpyDeviceToDevice);

        pyr_offset += pyr_w[l];
    }
    checkCUDAError("Fill in pict buffers");

    // Fill in derivatives, for the two pictures :
    fillDerivatives(gpu_img_pyramid_prev1,
                    gpu_array_deriv_x_0,
                    gpu_array_deriv_y_0);

    fillDerivatives(gpu_img_pyramid_prev2,
                    gpu_array_deriv_x_1,
                    gpu_array_deriv_y_1);

    checkCUDAError("Computing derivatives");

    hipMemset(gpu_status, 0, sizeof(char) * MAX_POINTS); // Not ready to track
    printf("CUDA : back pictures loaded %d x %d \n", w, h);
}


// Load current pair of pictures
// Called every time
void cudaLK::loadCurPictures(const IplImage *cur1,
                             const IplImage *cur2,
                             bool b_CvtToGrey) {

    if (!this->b_mem4_allocated) {
        printf("CUDA : error - memory must be allocated before use\n");
        return;
    } else if ( (cur1->width != w) || (cur1->height !=h) ) {
        printf("CUDA : error - pictures must have the same size\n");
        return;
    }

    int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
    int blocksH = h/_n_threads_y + ((h % _n_threads_y)?1:0);
    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Transfer from host memspace to gpu memspace
    if (b_CvtToGrey) {
        hipMemcpy2D (gpu_img_cur1_RGB, w*sizeof(uchar), cur1->imageData, cur1->widthStep, 3 * cur1->width * sizeof(uchar), cur1->height, hipMemcpyHostToDevice );
        hipMemcpy2D (gpu_img_cur2_RGB, w*sizeof(uchar), cur2->imageData, cur2->widthStep, 3 * cur2->width * sizeof(uchar), cur2->height, hipMemcpyHostToDevice );
    } else {
        hipMemcpy2D (gpu_img_cur1_RGB, w*sizeof(uchar), cur1->imageData, cur1->widthStep, cur1->width * sizeof(uchar), cur1->height, hipMemcpyHostToDevice );
        hipMemcpy2D (gpu_img_cur2_RGB, w*sizeof(uchar), cur2->imageData, cur2->widthStep, cur2->width * sizeof(uchar), cur2->height, hipMemcpyHostToDevice );
    }

    checkCUDAError("pictCopyToGPU");

    // Convert picture to floats & grey
    cvtPicture(true, b_CvtToGrey);
    checkCUDAError("pictConversion");

    // Build pyramids
    for(int i=0; i < _n_pyramids-1; i++) {
        kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_cur1[i], pyr_w[i], pyr_h[i], gpu_smoothed_cur1_x);
        kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_cur2[i], pyr_w[i], pyr_h[i], gpu_smoothed_cur2_x);

        kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_cur1_x, pyr_w[i], pyr_h[i], gpu_smoothed_cur1);
        kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_cur2_x, pyr_w[i], pyr_h[i], gpu_smoothed_cur2);

        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_cur1, pyr_w[i], pyr_h[i], gpu_img_pyramid_cur1[i+1], pyr_w[i+1], pyr_h[i+1]);
        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_cur2, pyr_w[i], pyr_h[i], gpu_img_pyramid_cur2[i+1], pyr_w[i+1], pyr_h[i+1]);
    }

    // Load hipArray buffer from pyramids
    int pyr_offset = 0;
    for (int l=0; l<_n_pyramids; ++l) {
        hipMemcpy2DToArrayAsync (gpu_array_pict_2,
                                  pyr_offset * sizeof(float),
                                  0,
                                  gpu_img_pyramid_cur2[l],
                                  sizeof(float)*pyr_w[l],
                                  sizeof(float)*pyr_w[l],
                                  pyr_h[l],
                                  hipMemcpyDeviceToDevice);

        hipMemcpy2DToArrayAsync(gpu_array_pict_3,
                                 pyr_offset * sizeof(float),
                                 0,
                                 gpu_img_pyramid_cur1[l],
                                 sizeof(float)*pyr_w[l],
                                 sizeof(float)*pyr_w[l],
                                 pyr_h[l],
                                 hipMemcpyDeviceToDevice);

        pyr_offset += pyr_w[l];
    }

    // Fill in derivatives, for the two pictures :
    fillDerivatives(gpu_img_pyramid_cur1,
                    gpu_array_deriv_x_3,
                    gpu_array_deriv_y_3);

    fillDerivatives(gpu_img_pyramid_cur2,
                    gpu_array_deriv_x_2,
                    gpu_array_deriv_y_2);

    hipMemset(gpu_status, 1, sizeof(char) * MAX_POINTS); // Ready to track

    checkCUDAError("Pyramid building");
}

void cudaLK::processTracking( int nPoints )
{


    // 2D-indexing for kernels
    int n_pts_ceil = MIN(nPoints, MAX_POINTS);
    int n_pts_sq = (int) round( sqrt(n_pts_ceil)) + 1;

    int blocksW = n_pts_sq/_n_threads_x +
                  ((n_pts_sq % _n_threads_x)?1:0);

    int blocksH = n_pts_sq/_n_threads_y +
                  ((n_pts_sq % _n_threads_y )?1:0);

    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    for( int l = _n_pyramids-1; l >= 0; l-- )
    {
        // Set constant parameters
        setSymbols(l);

        // Compute gradient descent parameters
        compute_spatial_grad <<<blocks, threads>>>(gpu_pt_indexes,
                                                   gpu_status,
                                                   gpu_neighbourhood_det,
                                                   gpu_neighbourhood_Iyy,
                                                   gpu_neighbourhood_Ixy,
                                                   gpu_neighbourhood_Ixx);

        // Compute the new position of the points
        track_pts_slim<<<blocks, threads>>>(gpu_pt_indexes,
                                            gpu_dx,
                                            gpu_dy,
                                            gpu_status,
                                            gpu_neighbourhood_det,
                                            gpu_neighbourhood_Iyy,
                                            gpu_neighbourhood_Ixy,
                                            gpu_neighbourhood_Ixx);
    }
}

void cudaLK::resetDisplacements() {
    hipMemset(gpu_dx,0, sizeof(float) * MAX_POINTS);
    hipMemset(gpu_dy,0, sizeof(float) * MAX_POINTS);
}

void cudaLK::releaseMem() {
    // Redundant tracking allocated
    if (this->b_mem4_allocated ) {
        printf("CudaLK : Releasing 4-Frames buffers\n");

        // Release pyramids
        for(int i=1; i < _n_pyramids; i++) {
            hipFree(gpu_img_pyramid_prev1[i]);
            hipFree(gpu_img_pyramid_prev2[i]);
            hipFree(gpu_img_pyramid_cur1[i]);
            hipFree(gpu_img_pyramid_cur2[i]);
        }

        // Release picture buffers
        hipFree(gpu_img_prev1_RGB);
        hipFree(gpu_img_prev2_RGB);
        hipFree(gpu_img_cur1_RGB);
        hipFree(gpu_img_cur2_RGB);

        // Release Pyramids
        hipFree(gpu_img_pyramid_prev1[0]);
        hipFree(gpu_img_pyramid_prev2[0]);
        hipFree(gpu_img_pyramid_cur1[0]);
        hipFree(gpu_img_pyramid_cur2[0]);

        // Release smoothed pictures (for pyramids)
        hipFree(gpu_smoothed_prev1_x);
        hipFree(gpu_smoothed_prev2_x);
        hipFree(gpu_smoothed_cur1_x);
        hipFree(gpu_smoothed_cur2_x);

        hipFree(gpu_smoothed_prev1);
        hipFree(gpu_smoothed_prev2);
        hipFree(gpu_smoothed_cur1);
        hipFree(gpu_smoothed_cur2);


        // Release spatial derivatives
        hipFree(gpu_deriv_x);
        hipFree(gpu_deriv_y);

        // Release compute intermediates
        hipFree(gpu_neighbourhood_det);
        hipFree(gpu_neighbourhood_Iyy);
        hipFree(gpu_neighbourhood_Ixy);
        hipFree(gpu_neighbourhood_Ixx);

        // Indexes
        hipFree ((void**) &gpu_pt_indexes);

        // Unbind textures
        hipUnbindTexture (gpu_textr_pict_0);
        hipUnbindTexture (gpu_textr_pict_1);

        hipUnbindTexture (gpu_textr_deriv_x);
        hipUnbindTexture (gpu_textr_deriv_y);

        // Release Arrays behind textures
        hipFreeArray (gpu_array_pict_0);
        hipFreeArray (gpu_array_pict_1);
        hipFreeArray (gpu_array_pict_2);
        hipFreeArray (gpu_array_pict_3);

        hipFreeArray (gpu_array_deriv_x_0);
        hipFreeArray (gpu_array_deriv_y_0);
        hipFreeArray (gpu_array_deriv_x_1);
        hipFreeArray (gpu_array_deriv_y_1);
        hipFreeArray (gpu_array_deriv_x_2);
        hipFreeArray (gpu_array_deriv_y_2);
        hipFreeArray (gpu_array_deriv_x_3);
        hipFreeArray (gpu_array_deriv_y_3);

        // Release Displacements
        hipFree(gpu_dx);
        hipFree(gpu_dy);

        hipFree(gpu_dx1);
        hipFree(gpu_dy1);

        hipFree(gpu_dx2);
        hipFree(gpu_dy2);

        hipFree(gpu_dx3);
        hipFree(gpu_dy3);

        hipHostFree(dx1);
        hipHostFree(dy1);
        hipHostFree(dx2);
        hipHostFree(dy2);
        hipHostFree(dx3);
        hipHostFree(dy3);
        hipHostFree(dx4);
        hipHostFree(dy4);
        hipHostFree(status);

        // Check GPU status
        hipFree(gpu_status);

        printf("CudaLK : buffers released\n");
    }

    // Simple tracking allocated
    if (this->b_mem_allocated) {
        // Free arrays
        for(int i=0; i < _n_pyramids; i++) {
            hipFree(gpu_img_pyramid_prev1[i]);
            hipFree(gpu_img_pyramid_cur1[i]);
        }

        hipFree(gpu_smoothed_prev_x);
        hipFree(gpu_smoothed_cur_x);
        hipFree(gpu_smoothed_prev);
        hipFree(gpu_smoothed_cur);
        hipFree(gpu_pt_indexes);

        hipHostFree(dx1);
        hipHostFree(dy1);

        hipFree(gpu_dx);
        hipFree(gpu_dy);
        hipFree(gpu_status);

        // Free textures
        hipFreeArray(gpu_array_pyramid_prev);
        hipFreeArray(gpu_array_pyramid_prev_Ix);
        hipFreeArray(gpu_array_pyramid_prev_Iy);
        hipFreeArray(gpu_array_pyramid_cur);

        delete [] status;
    }
}

// Coherent sparse tracking on stereo pair
//!\\ Previous set of pictures must be loaded prior to using this function
void cudaLK::run4Frames( IplImage  *cur1,
                         IplImage  *cur2,
                         float     *pt_to_track,
                         int       nPoints,
                         bool      cvtToGrey )
{
    int const & width  = cur1->width;
    int const & height = cur1->height;
    int const n_pts_ceil = MIN(nPoints, MAX_POINTS);

    // Check memory allocation before proceeding
    if (!b_mem4_allocated)
    {
        fprintf(stderr, "run4Frames : error - memory must be allocated and \n .. initial pictures loaded\n");
        exit(EXIT_FAILURE);
    }
    else if ((width != w) || (height != h))
    {
        fprintf(stderr, "run4Frames : error - Pictures must have the same size\n");
        exit(EXIT_FAILURE);
    }

    int win_size_full = _patch_radius;
    int win_size_short = 2;

    // Load current pictures & build pyramids
    loadCurPictures(cur1, cur2, cvtToGrey);

    // Load the coordinates of the points to track & define some settings
    hipMemcpy(gpu_pt_indexes, pt_to_track, 2 * n_pts_ceil * sizeof(float), hipMemcpyHostToDevice);

    checkCUDAError ("Loading pictures");

    // -----------------------------------------------------
    // "Loop" tracking
    // -----------------------------------------------------

    // -----------------------------------------------------
    // --- Step 1 -----

    // Bind textures and arrays...
    hipMemcpyToSymbol(HIP_SYMBOL(LK_win_size), &win_size_short, sizeof(int)); // win_size_short
    bindTextureUnits(gpu_array_pict_0,
                     gpu_array_pict_3,
                     gpu_array_deriv_x_0,
                     gpu_array_deriv_y_0);

    // Process
    resetDisplacements();
    processTracking(nPoints);
    checkCUDAError ("First step");

    // Copy back results
    hipMemcpy(gpu_dx1, gpu_dx, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice); // Handle "status" array
    hipMemcpy(gpu_dy1, gpu_dy, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice);

    hipDeviceSynchronize();

    hipMemcpyAsync(dx1, gpu_dx1, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost); // Non-blocking memcpy
    hipMemcpyAsync(dy1, gpu_dy1, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost);

    // -----------------------------------------------------
    // --- Step 2 -----

    // Change texture binding
    hipMemcpyToSymbol(HIP_SYMBOL(LK_win_size), &win_size_full, sizeof(int));
    bindTextureUnits(gpu_array_pict_3,
                     gpu_array_pict_2,
                     gpu_array_deriv_x_3,
                     gpu_array_deriv_y_3);

    // Process
    resetDisplacements();
    processTracking(nPoints);
    checkCUDAError ("Second step");

    // Copy back results
    hipMemcpy(gpu_dx2, gpu_dx, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice); // Handle "status" array
    hipMemcpy(gpu_dy2, gpu_dy, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice);

    hipDeviceSynchronize();

    hipMemcpyAsync(dx2, gpu_dx2, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost); // Non-blocking memcpy
    hipMemcpyAsync(dy2, gpu_dy2, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost);


    // -----------------------------------------------------
    // --- Step 3 -----
    hipMemcpyToSymbol(HIP_SYMBOL(LK_win_size), &win_size_short, sizeof(int)); // win_size_short

    // Change texture binding
    bindTextureUnits(gpu_array_pict_2,
                     gpu_array_pict_1,
                     gpu_array_deriv_x_2,
                     gpu_array_deriv_y_2);

    // Process
    resetDisplacements();
    processTracking(nPoints);
    checkCUDAError ("Third step");

    // Copy back results
    hipMemcpy(gpu_dx3, gpu_dx, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice);
    hipMemcpy(gpu_dy3, gpu_dy, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToDevice);

    hipDeviceSynchronize();

    hipMemcpyAsync(dx3, gpu_dx3, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost,0);
    hipMemcpyAsync(dy3, gpu_dy3, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost,0);


    // -----------------------------------------------------
    // --- Step 4 -----

    hipMemcpyToSymbol(HIP_SYMBOL(LK_win_size), &win_size_full, sizeof(int));

    // Change texture binding
    bindTextureUnits(gpu_array_pict_1,
                     gpu_array_pict_0,
                     gpu_array_deriv_x_1,
                     gpu_array_deriv_y_1);

    // Process
    resetDisplacements();
    processTracking(nPoints);

    hipDeviceSynchronize();
    checkCUDAError ("Last step");

    // Copy back results to Host (non blocking memcpy to pinned memory)
    hipMemcpyAsync(dx4, gpu_dx, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost);
    hipMemcpyAsync(dy4, gpu_dy, sizeof(float) * n_pts_ceil, hipMemcpyDeviceToHost);
    hipMemcpyAsync(status, gpu_status, sizeof(char) * n_pts_ceil, hipMemcpyDeviceToHost);

    // Cpy new point position :
    hipMemcpy(pt_to_track, gpu_pt_indexes, sizeof(float) * 2 * n_pts_ceil, hipMemcpyDeviceToHost);

    // Swap the pictures & pyramids (current -> back)
    swapPyramids();
}


void cudaLK::setSymbols(int pyr_level) {
    int pyr_deriv_offset = 0;
    char initGuess;

    for (int i=0; i<pyr_level; ++i) {
        pyr_deriv_offset += pyr_w[i];
    }
    hipMemcpyToSymbol(HIP_SYMBOL(LK_width_offset), &pyr_deriv_offset, sizeof(int));

    initGuess = (pyr_level == _n_pyramids-1);
    hipMemcpyToSymbol(HIP_SYMBOL(LK_pyr_w),     &pyr_w[pyr_level],    sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LK_pyr_h),     &pyr_h[pyr_level],    sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LK_pyr_level), &pyr_level,           sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LK_scaling),   &scaling[pyr_level],  sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(LK_init_guess),&initGuess,           sizeof(char));
}


void cudaLK::setupTextures() {
    // Picture buffers
    gpu_textr_pict_0.normalized = 0;
    gpu_textr_pict_0.filterMode = hipFilterModeLinear;
    gpu_textr_pict_0.addressMode[0] = hipAddressModeClamp;  // Handle request outside boundaries
    gpu_textr_pict_0.addressMode[1] = hipAddressModeClamp;

    gpu_textr_pict_1.normalized = 0;
    gpu_textr_pict_1.filterMode = hipFilterModeLinear;
    gpu_textr_pict_1.addressMode[0] = hipAddressModeClamp;  // Handle request outside boundaries
    gpu_textr_pict_1.addressMode[1] = hipAddressModeClamp;

    // Spatial derivatives
    gpu_textr_deriv_x.normalized = 0;
    gpu_textr_deriv_x.filterMode = hipFilterModeLinear;
    gpu_textr_deriv_x.addressMode[0] = hipAddressModeClamp;  // Handle request outside boundaries
    gpu_textr_deriv_x.addressMode[1] = hipAddressModeClamp;

    gpu_textr_deriv_y.normalized = 0;
    gpu_textr_deriv_y.filterMode = hipFilterModeLinear;
    gpu_textr_deriv_y.addressMode[0] = hipAddressModeClamp;  // Handle request outside boundaries
    gpu_textr_deriv_y.addressMode[1] = hipAddressModeClamp;

    checkCUDAError ("Initializing textures");
}

void cudaLK::sobelFiltering(const float *pict_in,
                            const int w,
                            const int h,
                            float *pict_out) {
    // TODO

    // 2D-indexing for kernel managements
    int _n_threads_x = NTHREAD_X;
    int _n_threads_y = NTHREAD_Y;

    int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
    int blocksH = h/_n_threads_y + ((h % _n_threads_y)?1:0);

    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Compute via kernels
    kernelSobelX <<<blocks, threads>>>(pict_in, w, h, buff1);

    kernelSobelY <<<blocks, threads>>>(pict_in, w, h, buff2);

    // Mix sobel gradient into one picture
    kernelAdd <<<blocks, threads>>>(buff1, buff2, w, h, pict_out);

    hipDeviceSynchronize();
}

void cudaLK::sobelFilteringX(const float *pict_in,
                             const int w,
                             const int h,
                             float *pict_out) {
    // TODO

    // 2D-indexing for kernel managements
    int _n_threads_x = NTHREAD_X;
    int _n_threads_y = NTHREAD_Y;

    int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
    int blocksH = h/_n_threads_y + ((h % _n_threads_y)?1:0);

    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Compute via kernels
    kernelSobelX <<<blocks, threads>>>(pict_in, w, h, pict_out);
    hipDeviceSynchronize();
}

void cudaLK::sobelFilteringY(const float *pict_in,
                             const int w,
                             const int h,
                             float *pict_out) {
    // TODO

    // 2D-indexing for kernel managements
    int _n_threads_x = NTHREAD_X;
    int _n_threads_y = NTHREAD_Y;

    int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
    int blocksH = h/_n_threads_y + ((h % _n_threads_y)?1:0);

    dim3 blocks(blocksW, blocksH);
    dim3 threads(_n_threads_x, _n_threads_y);

    // Compute via kernels
    kernelSobelY <<<blocks, threads>>>(pict_in, w, h, pict_out);
    hipDeviceSynchronize();
}

// Swap current/backbuffer pyramids
void cudaLK::swapPyramids () {
    // Swap pyramid arrays:
    hipMemcpy2DArrayToArray  (gpu_array_pict_0,
                               0, 0,
                               gpu_array_pict_3,
                               0, 0,
                               2 * w * sizeof(float),
                               h,
                               hipMemcpyDeviceToDevice);

    hipMemcpy2DArrayToArray  (gpu_array_pict_1,
                               0, 0,
                               gpu_array_pict_2,
                               0, 0,
                               2 * w * sizeof(float),
                               h,
                               hipMemcpyDeviceToDevice);


    // Swap derivatives pyramid :
    hipMemcpy2DArrayToArray (gpu_array_deriv_x_0,
                              0, 0,
                              gpu_array_deriv_x_3,
                              0, 0,
                              2 * w * sizeof(float),
                              h,
                              hipMemcpyDeviceToDevice);

    hipMemcpy2DArrayToArray (gpu_array_deriv_y_0,
                              0, 0,
                              gpu_array_deriv_y_3,
                              0, 0,
                              2 * w * sizeof(float),
                              h,
                              hipMemcpyDeviceToDevice);

    hipMemcpy2DArrayToArray (gpu_array_deriv_x_1,
                              0, 0,
                              gpu_array_deriv_x_2,
                              0, 0,
                              2 * w * sizeof(float),
                              h,
                              hipMemcpyDeviceToDevice);

    hipMemcpy2DArrayToArray (gpu_array_deriv_y_1,
                              0, 0,
                              gpu_array_deriv_y_2,
                              0, 0,
                              2 * w * sizeof(float),
                              h,
                              hipMemcpyDeviceToDevice);
}
/********************************
 * Previous work....
 *********************************/

/*
 *void cudaLK::runTracking(unsigned char *prev,     // Previous picture
                         unsigned char *cur,      // New picture
                         int _w,                  // Picture width
                         int _h,                  // Picture height
                         float *pt_to_track,      // 2D array of indexes : points to track (floats !)
                         int n_pts,               // Number of points to track
                         bool b_CvtToGrey)        // Do the RGB2GRAY conversion or not (single channel picture)
{
  //
  // Sparse optical field calculus : follow points specified in "pt_to_track" array
  //

  if (! b_mem_allocated) {
    w = _w;
    h = _h;
    initMem();
  }

  int _n_threads_x = NTHREAD_X;
  int _n_threads_y = NTHREAD_Y;

  int blocksW = w/_n_threads_x + ((w % _n_threads_x)?1:0);
  int blocksH = h/_n_threads_y + ((h % _n_threads_y )?1:0);
  dim3 blocks(blocksW, blocksH);
  dim3 threads(_n_threads_x, _n_threads_y);
  int blocks1D = (w*h)/256 + (w*h % 256?1:0); // for greyscale
  int blocks1D_tracking = n_pts/256 + (n_pts % 256 ? 1:0);

  // Copy image to GPU :
  if(b_CvtToGrey) {
    hipMemcpy(gpu_img_prev_RGB, prev, w*h*3, hipMemcpyHostToDevice);
    hipMemcpy(gpu_img_cur_RGB, cur, w*h*3, hipMemcpyHostToDevice);
    checkCUDAError("start");
  } else {
    hipMemcpy(gpu_img_prev_RGB, prev, w*h, hipMemcpyHostToDevice);
    hipMemcpy(gpu_img_cur_RGB, cur, w*h, hipMemcpyHostToDevice);
    checkCUDAError("start");
  }

  // Copy indexes to follow to GPU :
  hipMemcpy(gpu_pt_indexes, pt_to_track, 2*n_pts*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError ("Copying indexes to follow");


  // Convert pictures (float & grey)
  if (b_CvtToGrey) {
    // RGB -> grey
    convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_prev_RGB, gpu_img_pyramid_prev1[0], w*h);
    convertRGBToGrey<<<blocks1D, 256>>>(gpu_img_cur_RGB, gpu_img_pyramid_cur1[0], w*h);
    hipDeviceSynchronize();
    checkCUDAError("convertRGBToGrey");
  } else {
    // Simply convert char to float in kernel
    convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_prev_RGB, gpu_img_pyramid_prev1[0], w*h);
    convertGreyToFloat<<<blocks1D, 256>>>(gpu_img_cur_RGB, gpu_img_pyramid_cur1[0], w*h);
    hipDeviceSynchronize();
    checkCUDAError("convertToFloat");
  }

  // Build pyramids
  for(int i=0; i < _n_pyramids-1; i++) {
    kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_prev1[i], pyr_w[i], pyr_h[i], gpu_smoothed_prev_x);
    kernelSmoothX<<<blocks, threads>>>(gpu_img_pyramid_cur1[i], pyr_w[i], pyr_h[i], gpu_smoothed_cur_x);
    hipDeviceSynchronize();
    kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_prev_x, pyr_w[i], pyr_h[i], gpu_smoothed_prev);
    kernelSmoothY<<<blocks, threads>>>(gpu_smoothed_cur_x, pyr_w[i], pyr_h[i], gpu_smoothed_cur);
    hipDeviceSynchronize();

    pyrDownsample<<<blocks, threads>>>(gpu_smoothed_prev, pyr_w[i], pyr_h[i], gpu_img_pyramid_prev1[i+1], pyr_w[i+1], pyr_h[i+1]);
    pyrDownsample<<<blocks, threads>>>(gpu_smoothed_cur,  pyr_w[i], pyr_h[i], gpu_img_pyramid_cur1[i+1],  pyr_w[i+1], pyr_h[i+1]);
    hipDeviceSynchronize();

    checkCUDAError("pyrDownsample here");
  }

  hipMemset(gpu_status, 1, sizeof(char) * MAX_POINTS);

  // Do the actual tracking
  for(int l=_n_pyramids-1; l >= 0; l--) {

    // Copy arrays to texture
    hipMemcpy2DToArray(gpu_array_pyramid_prev, 0, 0, gpu_img_pyramid_prev1[l],
                        sizeof(float)*pyr_w[l], sizeof(float)*pyr_w[l], pyr_h[l], hipMemcpyDeviceToDevice);

    hipMemcpy2DToArray(gpu_array_pyramid_cur, 0, 0, gpu_img_pyramid_cur1[l],
                        sizeof(float)*pyr_w[l], sizeof(float)*pyr_w[l], pyr_h[l], hipMemcpyDeviceToDevice);

    // Track
    //    trackPt<<<blocks1D_tracking, 256>>>(gpu_pt_indexes, n_pts, w, h, pyr_w[l], pyr_h[l], scaling[l], l, (l == levels-1), gpu_dx, gpu_dy, 10, .3f, gpu_status);
    trackPt<<<blocks1D_tracking, 256>>>(gpu_pt_indexes, n_pts, w, h, pyr_w[l], pyr_w[l], scaling[l], l, (l == _n_pyramids-1), gpu_dx, gpu_dy, 10, .3f, gpu_status);

    hipDeviceSynchronize();
  }

  // Copy back results
  hipMemcpy(dx1, gpu_dx, sizeof(float)*n_pts, hipMemcpyDeviceToHost);
  hipMemcpy(dy1, gpu_dy, sizeof(float)*n_pts, hipMemcpyDeviceToHost);
  hipMemcpy(status, gpu_status, sizeof(char)*n_pts, hipMemcpyDeviceToHost);
}
*/
